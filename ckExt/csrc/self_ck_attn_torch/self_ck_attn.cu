#include <iostream>
#include <math.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "fused_attention.hpp"

namespace ck_attn {
namespace self {

std::vector<torch::Tensor> fwd_cuda(torch::Tensor const &query,
                                    torch::Tensor const &key,
                                    torch::Tensor const &value,
                                    torch::Tensor const &out,
                                    float dropout_prob, const int best_op_id) {

  const int sequences = query.size(0);
  const int heads = query.size(1);
  const int seq_len = query.size(2);
  const int head_dim = query.size(3);


  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  auto act_options = query.options().requires_grad(false);


  void *query_ptr = static_cast<void *>(query.data_ptr());
  void *key_ptr = static_cast<void *>(key.data_ptr());
  void *value_ptr = static_cast<void *>(value.data_ptr());
  void *out_ptr = static_cast<void *>(out.data_ptr());

  torch::Tensor attn_outputs = 
      //torch::empty({sequences, seq_len, heads, head_dim});
      torch::empty({sequences, seq_len, heads, head_dim}, act_options);
  void *attn_outputs_ptr = static_cast<void *>(attn_outputs.data_ptr());

  //fused_attention(sequences, heads, seq_len, seq_len, head_dim, head_dim, query_ptr, key_ptr, value_ptr, attn_outputs_ptr, best_op_id);
  fused_attention(sequences, heads, seq_len, seq_len, head_dim, head_dim, query_ptr, key_ptr, value_ptr, out_ptr, best_op_id);
  //fused_attention(sequences, heads, seq_len, seq_len, head_dim, head_dim, query.data_ptr<half>(), key.data_ptr<half>(), value.data_ptr<half>(), attn_outputs.data_ptr<half>(), best_op_id);


  return { attn_outputs };

}

} // end namespace self
} // end namespace multihead_attn
